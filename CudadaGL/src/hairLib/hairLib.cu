#include "hip/hip_runtime.h"
#include "hairLib.h"
#include <fstream>

__global__ void initHairs(float* X, float* Y, float*Z, float hxy, float hz)
{
	int line = blockIdx.x;
	int Zi = blockIdx.y;

	int lineOffset = line * blockDim.x * blockDim.y;
	int Zoffset = Zi * (gridDim.x) * blockDim.x * blockDim.y;
	
	int idx = Zoffset + lineOffset + threadIdx.x ;
	X[idx] = threadIdx.x * hxy;
	Y[idx] = line * hxy;
	Z[idx] = (Zi-1) * hz * (Zi > 1); // Two first position are in Z = 0;

	//int slice = blockIdx.y;
	//int Zi = threadIdx.x;
	//int hairIdInLine = blockIdx.x * blockDim.y + threadIdx.y;

	//int idx = slice * blockDim.x * blockDim.y * gridDim.x + hairIdInLine * blockDim.x + threadIdx.x;

	//X[idx] = hairIdInLine * hxy;
	//Y[idx] = slice * hxy;
	//Z[idx] = Zi * hz;
}

__global__ void applyGravity(float* X, float* Y, float*Z,
							 float* vx, float* vy, float* vz,
							 float dt)
{
	const float gravity = 9.81;
	const float mass = 1e-5;
	const float massInv = 1e5;
	const float alpha = 0.5 * mass; // Rayleigh damping


	int line = blockIdx.x;
	int Zi = blockIdx.y;

	int lineOffset = line * blockDim.x * blockDim.y;
	int Zoffset = Zi * (gridDim.x) * blockDim.x * blockDim.y;
	
	int idx = Zoffset + lineOffset + threadIdx.x;


	float Imass = -gravity * dt / massInv;
	float Idamp = - alpha * mass * dt;
	
	vy[idx] += (Imass + Idamp) / mass * (Zi > 0);
}

__global__ void integrateK(float* X, float* Y, float*Z,
						  float* vx, float* vy, float* vz,
						  float dt)
{
	int line = blockIdx.x;
	int Zi = blockIdx.y;

	int lineOffset = line * blockDim.x * blockDim.y;
	int Zoffset = Zi * (gridDim.x) * blockDim.x * blockDim.y;
	
	int idx = Zoffset + lineOffset + threadIdx.x ;

	// Integrate velocity
	X[idx] += vx[idx] * dt;
	Y[idx] += vy[idx] * dt;
	Z[idx] += vz[idx] * dt;
}

__global__ void applyConstraint(float* X, float* Y, float*Z,
							 float* vx, float* vy, float* vz,
							 float hz,
							 float dt)
{
	const float mass = 1e-5;
	const float massInv = 1e5;

	__shared__ float  massPositionX[256*2];
	__shared__ float  massPositionY[256*2];
	__shared__ float  massPositionZ[256*2];

	__shared__ float  massVelocityX[256*2];
	__shared__ float  massVelocityY[256*2];
	__shared__ float  massVelocityZ[256*2];


	int line = blockIdx.x;
	int lineOffset = line * blockDim.x * blockDim.y;
	
	int depth = 30;
	for(int z = 0 ; z < depth-1 ; ++z)
	{
		int ZoffC = z * (gridDim.x) * blockDim.x * blockDim.y;
		int ZoffN = (z+1) * (gridDim.x) * blockDim.x * blockDim.y;


		int idxC = ZoffC + lineOffset + threadIdx.x ;
		int idxN = ZoffN + lineOffset + threadIdx.x ;


		massPositionX[threadIdx.x] = X[idxC];
		massPositionY[threadIdx.x] = Y[idxC];
		massPositionZ[threadIdx.x] = Z[idxC];

		massVelocityX[threadIdx.x] = vx[idxC];
		massVelocityY[threadIdx.x] = vy[idxC];
		massVelocityZ[threadIdx.x] = vz[idxC];

		__syncthreads();
		
		massPositionX[256 + threadIdx.x] = X[idxN];
		massPositionY[256 + threadIdx.x] = Y[idxN];
		massPositionZ[256 + threadIdx.x] = Z[idxN];

		massVelocityX[256 + threadIdx.x] = vx[idxN];
		massVelocityY[256 + threadIdx.x] = vy[idxN];
		massVelocityZ[256 + threadIdx.x] = vz[idxN];

		__syncthreads();

		const float relX = massPositionX[256 + threadIdx.x] - massPositionX[threadIdx.x];
		const float relY = massPositionY[256 + threadIdx.x] - massPositionY[threadIdx.x];
		const float relZ = massPositionZ[256 + threadIdx.x] - massPositionZ[threadIdx.x];


		const float relvX = massVelocityX[256 + threadIdx.x] - massVelocityX[threadIdx.x];
		const float relvY = massVelocityY[256 + threadIdx.x] - massVelocityY[threadIdx.x];
		const float relvZ = massVelocityZ[256 + threadIdx.x] - massVelocityZ[threadIdx.x];


		float dist = sqrt( relX * relX + relY * relY + relZ * relZ);
		if(dist != 0.)
		{
			const float dx = relX / dist;
			const float dy = relY / dist;
			const float dz = relZ / dist;

			const float velProj = dx * relvX + dy * relvY + dz * relvZ;
			const float gap = dist - hz * (z > 0);

			const float constI = (gap / dt + velProj) / (massInv + massInv * (z > 0));

			vx[idxC] += constI * dx / mass*(z > 0); 
			vy[idxC] += constI * dy / mass*(z > 0);
			vz[idxC] += constI * dz / mass*(z > 0);

			vx[idxN] += -constI * dx / mass; 
			vy[idxN] += -constI * dy / mass;
			vz[idxN] += -constI * dz / mass;
		}
	}
	//dist = self.mass2.pos - self.mass1.pos;
	//if linalg.norm(dist) != 0.:
	//distUnit = dist / linalg.norm(dist);

	//relVel = dot((self.mass2.vel - self.mass1.vel), distUnit);
	//relDist = linalg.norm(dist) - self.refDist;

	//I = (relDist / dt + relVel) / (self.mass1.iMass + self.mass2.iMass) * distUnit;
	//self.mass1.applyImpulse(I);
	//self.mass2.applyImpulse(-I);
}

HairSimulation::HairSimulation(float x, float y, float z, float radius, int nbPartByHair, float hxy, float hz) :
x(x),
y(y), 
z(z),
radius(radius),
nbHairs(nbHairs),
blockX(256),
blockY(1),
d_x(NULL),
d_y(NULL),
d_z(NULL),
hxy(hxy),
hz(hz)
{
	
}

HairSimulation::~HairSimulation()
{
	if(d_x)
		hipFree(d_x);
	if(d_y)
		hipFree(d_y);
	if(d_z)
		hipFree(d_z);
}

void HairSimulation::initHair()
{
	dim3 gridSize(256, 30); 
	dim3 blockSize(blockX, blockY);

	int size = gridSize.x * gridSize.y * blockSize.x * blockSize.y;


	/////////////////////////////////////////
	//            Init Position            //
	/////////////////////////////////////////
	hipMalloc(&d_x, size * sizeof(float));
	hipMalloc(&d_y, size * sizeof(float));
	hipMalloc(&d_z, size * sizeof(float));


	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord( start, 0 );

	initHairs<<<gridSize, blockSize >>>(d_x, d_y, d_z, hxy, hz);
	
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );

	std::ofstream out("log.cudada");
	out << "initHairs : " << time << std::endl;
	out.close();

	X.resize(size);
	Y.resize(size);
	Z.resize(size);

	hipMemcpy(&(X[0]), d_x, size * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(&(Y[0]), d_y, size * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(&(Z[0]), d_z, size * sizeof(float), hipMemcpyDeviceToHost);
	/////////////////////////////////////////
	//            Init Velocity            //
	/////////////////////////////////////////
	hipMalloc(&d_vx, size * sizeof(float));
	hipMalloc(&d_vy, size * sizeof(float));
	hipMalloc(&d_vz, size * sizeof(float));

	hipMemset(d_vx, 0, sizeof(float));
	hipMemset(d_vy, 0, sizeof(float));
	hipMemset(d_vz, 0, sizeof(float));


}

void HairSimulation::integrate(float dt)
{
	dim3 gridSize(256, 30);
	dim3 blockSize(blockX, blockY);
	int size = gridSize.x * gridSize.y * blockSize.x * blockSize.y;

	/////////////////////////////////////////
	//        Integrate Free Motion        //
	/////////////////////////////////////////
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord( start, 0 );

	applyGravity<<<gridSize, blockSize >>>(d_x, d_y, d_z, 
		d_vx, d_vy, d_vz,
		dt);
	
	
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );

	std::ofstream out("log.cudada", std::ios::app);
	out << "gravity : " << time << std::endl;
	

	/////////////////////////////////////////
	//   Integrate Constrained Motion      //
	/////////////////////////////////////////
	dim3 gridSize2(256, 1);
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord( start, 0 );

	applyConstraint<<<gridSize2, blockSize >>>(d_x, d_y, d_z, 
		d_vx, d_vy, d_vz,
		hz,
		dt);

	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );
	out << hipGetErrorString(hipGetLastError()) << std::endl;
	out << "constraint : " << time << std::endl;

	/////////////////////////////////////////
	//            Time integration         //
	/////////////////////////////////////////
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord( start, 0 );
	integrateK<<<gridSize, blockSize >>>(d_x, d_y, d_z, 
		d_vx, d_vy, d_vz,
		dt);
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );

	out << "integrate : " << time << std::endl;
	out.close();
	/////////////////////////////////////////
	//             Transfter data          //
	/////////////////////////////////////////
	hipMemcpy(&(X[0]), d_x, size * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(&(Y[0]), d_y, size * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(&(Z[0]), d_z, size * sizeof(float), hipMemcpyDeviceToHost);
}


const std::vector<float>& HairSimulation::getMassPositionX()
{
	return X;
}


const std::vector<float>& HairSimulation::getMassPositionY()
{
	return Y;
}


const std::vector<float>& HairSimulation::getMassPositionZ()
{
	return Z;
}