#include "hip/hip_runtime.h"
#include "smokeLib.h"
#include <fstream>
#include <iostream>

#define M_PI 3.141516


__global__ void initSmokeK(float* density)
{
	int slice = blockIdx.y;
	int column = blockIdx.x;
	int row = threadIdx.x;

	int idx = slice * gridDim.x * gridDim.y + column * blockDim.x + row;
	density[idx] = column * 1.0 / blockDim.x;
}

__global__ void propagateLight(float* density, float* radiance)
{
	const float albedo = 1.0;
	const float light = 1.0;
	const float h = 0.1;

	__shared__ float colDens[8 * 256]; // load eight cols
	int slice = blockIdx.x;

	float tRay = 1.;
	for(int depth = 0 ; depth < 32; ++depth)
	{
		for(int passId = 0 ; passId < 8 ; ++passId)
		{
			int row = threadIdx.x % 8;
			int col = passId * 32 + threadIdx.x / 8;

			int gidx = slice * gridDim.x * gridDim.y + col * blockDim.x +  depth * 8 + row;
			int sidx = row * 256 + (passId * 32 + threadIdx.x / 8);
			colDens[sidx] = density[gidx];
		}

		//__syncthreads();

		for(int row = 0 ; row < 8 ; ++row)
		{
			int col = threadIdx.x;
			const float tVox = exp(-0.3 * h * colDens[row * blockDim.x + col]);

			tRay *= tVox;
			colDens[row * blockDim.x + col] = albedo * light * tRay ;
			
		}

		//__syncthreads();

		for(int passId = 0 ; passId < 8 ; ++passId)
		{
			int row = threadIdx.x % 8;
			int col = passId * 32 + threadIdx.x / 8;

			int gidx = slice * gridDim.x * gridDim.y + col * blockDim.x + depth * 8 + row;
			int sidx = row * 256 + (passId * 32 + threadIdx.x / 8);
			radiance[gidx] = colDens[sidx];
		}
	}
}


__global__ void renderImage(float* density, float* radiance, float* image)
{
	const float albedo = 1.0;
	const float light = 1.0;
	const float h = 0.1;

	__shared__ float colDens[8 * 256]; // load eight cols
	int slice = blockIdx.x;

	float pixel = 0.;
	float tRay = 1.;
	for(int depth = 0 ; depth < 32; ++depth)
	{
		for(int passId = 0 ; passId < 8 ; ++passId)
		{
			int row = threadIdx.x % 8;
			int col = passId * 32 + threadIdx.x / 8;

			int gidx = slice * gridDim.x * gridDim.y + col * blockDim.x +  depth * 8 + row;
			int sidx = row * 256 + (passId * 32 + threadIdx.x / 8);
			colDens[sidx] = density[gidx];
		}

		__syncthreads();

		for(int row = 0 ; row < 8 ; ++row)
		{
			int col = threadIdx.x;
			const float tVox = exp(-0.3 * h * colDens[row * blockDim.x + col]);

			tRay *= tVox;
			colDens[row * blockDim.x + col] = albedo * light * tRay ;
		}

		__syncthreads();

		for(int passId = 0 ; passId < 8 ; ++passId)
		{
			int row = threadIdx.x % 8;
			int col = passId * 32 + threadIdx.x / 8;

			int gidx = slice * gridDim.x * gridDim.y + col * blockDim.x + depth * 8 + row;
			int sidx = row * 256 + (passId * 32 + threadIdx.x / 8);
			radiance[gidx] = colDens[sidx];
		}
	}
}

SmokeRenderer::SmokeRenderer() :
d_density(NULL),
d_image(NULL),
gridX(256),
gridY(256),
gridZ(256)
{
	
}

SmokeRenderer::~SmokeRenderer()
{
	if(d_density)
		hipFree(d_density);
	if(d_image)
		hipFree(d_image);
}

void SmokeRenderer::initSmoke()
{
	dim3 gridSize(gridY, gridX); 
	dim3 blockSize(gridZ);

	int size = gridX * gridY * gridZ;
	int imageSize = gridX * gridY;


	/////////////////////////////////////////
	//            Init Position            //
	/////////////////////////////////////////
	hipMalloc(&d_density, size * sizeof(float));
	hipMalloc(&d_radiance, size * sizeof(float));
	hipMalloc(&d_image, imageSize * sizeof(float));

	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord( start, 0 );

	initSmokeK<<<gridSize, blockSize >>>(d_density);
	
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );

	std::ofstream out("log.smoke");
	out << "initSmoke : " << time << std::endl;
	out.close();

	density.resize(size);
	radiance.resize(size);
	image.resize(imageSize);
	
	hipMemcpy(&(density[0]), d_density, size * sizeof(float), hipMemcpyDeviceToHost);
}


void SmokeRenderer::render()
{
	dim3 gridSize(gridX); 
	dim3 blockSize(gridY);
	int size = gridX * gridY * gridZ;

	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord( start, 0 );

	propagateLight<<<gridSize, blockSize >>>(d_density, d_radiance);

	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );


	std::ofstream out("log.smoke", std::ios::app);
	out << "render : " << time << std::endl;
	out.close();

	std::cout << hipGetErrorString(hipGetLastError()) << std::cout;


	hipMemcpy(&(radiance[0]), d_radiance, size * sizeof(float), hipMemcpyDeviceToHost);

}


const std::vector<float>& SmokeRenderer::getDensity()
{
	return density;
}


const std::vector<float>& SmokeRenderer::getImage()
{
	return image;
}


const std::vector<float>& SmokeRenderer::getRadiance()
{
	return radiance;
}