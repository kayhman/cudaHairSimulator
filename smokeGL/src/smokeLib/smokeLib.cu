#include "hip/hip_runtime.h"
#include "smokeLib.h"
#include <fstream>
#include <iostream>

#define M_PI 3.141516


__global__ void initSmokeK(float* density)
{
	int slice = blockIdx.y;
	int column = blockIdx.x;
	int row = threadIdx.x;

	int idx = slice * gridDim.x * gridDim.y + row * blockDim.x + column;
	density[idx] = column * 1.0 / blockDim.x;
}

__global__ void propagateLight(float* density, float* radiance,  float* image)
{
	float albedo = 1.0;
	float light = 1.0;
	float h = 0.1;

	int slice = blockIdx.x;
	int col = threadIdx.x;


	float tRay = 1.;
	for(int row = 0 ; row < 256 ; ++row)
	{
		int gidx = slice * gridDim.x * gridDim.y + row * blockDim.x + col;

		float tVox = exp(-0.3 * h * density[gidx]);
		tRay *= tVox;

		int ridx = slice * gridDim.x * gridDim.y + row + col * blockDim.x;
		radiance[gidx] = albedo * light * tRay ;
	}


	for(int row = 0 ; rcol < 256 ; ++row)
	{
		int gidx = slice * gridDim.x * gridDim.y + row * blockDim.x + col;

		float tVox = exp(-0.3 * h * density[gidx]);
		tRay *= tVox;

		int ridx = slice * gridDim.x * gridDim.y + row + col * blockDim.x;
		radiance[gidx] = albedo * light * tRay ;
	}

}

__global__ void renderImage(float* density, float* radiance, float* image)
{
}

SmokeRenderer::SmokeRenderer() :
d_density(NULL),
d_image(NULL),
gridX(256),
gridY(256),
gridZ(256)
{
	
}

SmokeRenderer::~SmokeRenderer()
{
	if(d_density)
		hipFree(d_density);
	if(d_image)
		hipFree(d_image);
}

void SmokeRenderer::initSmoke()
{
	dim3 gridSize(gridY, gridX); 
	dim3 blockSize(gridZ);

	int size = gridX * gridY * gridZ;
	int imageSize = gridX * gridY;


	/////////////////////////////////////////
	//            Init Position            //
	/////////////////////////////////////////
	hipMalloc(&d_density, size * sizeof(float));
	hipMalloc(&d_radiance, size * sizeof(float));
	hipMalloc(&d_image, imageSize * sizeof(float));

	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord( start, 0 );

	initSmokeK<<<gridSize, blockSize >>>(d_density);
	
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );

	std::ofstream out("log.smoke");
	out << "initSmoke : " << time << std::endl;
	out.close();

	density.resize(size);
	radiance.resize(size);
	image.resize(imageSize);
	
	hipMemcpy(&(density[0]), d_density, size * sizeof(float), hipMemcpyDeviceToHost);
}


void SmokeRenderer::render()
{
	dim3 gridSize(gridX); 
	dim3 blockSize(gridY);
	int size = gridX * gridY * gridZ;

	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord( start, 0 );

	propagateLight<<<gridSize, blockSize >>>(d_density, d_radiance);

	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );


	std::ofstream out("log.smoke", std::ios::app);
	out << "render : " << time << std::endl;
	out.close();

	std::cout << hipGetErrorString(hipGetLastError()) << std::cout;


	hipMemcpy(&(radiance[0]), d_radiance, size * sizeof(float), hipMemcpyDeviceToHost);

}


const std::vector<float>& SmokeRenderer::getDensity()
{
	return density;
}


const std::vector<float>& SmokeRenderer::getImage()
{
	return image;
}


const std::vector<float>& SmokeRenderer::getRadiance()
{
	return radiance;
}